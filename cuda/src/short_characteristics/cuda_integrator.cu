
#include <hip/hip_runtime.h>
﻿#ifdef USE_CUDA
//***********************************************************************//
//*********************Functions from device*****************************//
//***********************************************************************//
#include "cuda_integrator.h"
#include "global_def.h"

namespace c_dir = cuda::device::direction_integrator;

__device__ Type c_dir::Gamma(const Vector3 &direction, const Vector3 &direction2) {
  Type sum = direction.dot(direction2);
  return (3. * (1. + sum * sum)) / 4.;
}
__device__ Type c_dir::GammaT(const Vector3 &direction, const Vector3 &direction2) {
  Type cos = direction.dot(direction2);
  if (cos >= 0.9999999) {
    return 0;
  }
  Type cos2 = cos * cos;
  return (3. * (1. + cos2) * sqrt(1. - cos2)) / 8.;
}

__device__ Type c_dir::IntegrateByCell(const IdType num_cell, const geo::grid_directions_device_t *dir, const geo::grid_device_t *grid) {
  const IdType M = dir->size;
  const IdType N = grid->size;

  Type res = 0;
  for (IdType i = 0; i < M; i++) {
    IdType pos = CELL_SIZE * (N * i + num_cell);

    Type I = 0;
    for (IdType k = 0; k < CELL_SIZE; k++) {
      I += grid->illum[pos + k];
    }
    I /= CELL_SIZE;

    res += I * dir->directions[i].area;
  }

  return res / dir->full_area;
}

__device__ void c_dir::IntegrateByFaces3(const IdType num_cell, const geo::grid_directions_device_t *dir_grid, geo::grid_device_t *grid, Vector3 *Stream) {

  const IdType M = dir_grid->size;
  const IdType N = grid->size;

  for (int h = 0; h < CELL_SIZE; h++) {
    Stream[h] = Vector3::Zero();
  }

  for (IdType i = 0; i < M; i++) {
    IdType pos = CELL_SIZE * (N * i + num_cell);
    for (int f = 0; f < CELL_SIZE; f++) {
      Stream[f] += dir_grid->directions[i].dir * (grid->illum[pos + f] * dir_grid->directions[i].area);
    }
  }

  for (int h = 0; h < CELL_SIZE; h++)
    Stream[h] /= dir_grid->full_area;

  return;
}

__device__ void c_dir::IntegrateByFaces9(const IdType num_cell, const geo::grid_directions_device_t *dir_grid, geo::grid_device_t *grid, Matrix3 *Impuls) {
  const IdType M = dir_grid->size;
  const IdType N = grid->size;

  for (int h = 0; h < CELL_SIZE; h++) {
    Impuls[h] = Matrix3::Zero();
  }

  for (IdType dir = 0; dir < M; dir++) {
    IdType pos = CELL_SIZE * (N * dir + num_cell);
    for (int f = 0; f < CELL_SIZE; f++) {
      for (int i = 0; i < 3; i++)
        for (int k = 0; k < 3; k++) {
          Type I = grid->illum[pos + f];
          Impuls[f](i, k) += dir_grid->directions[dir].dir[i] * dir_grid->directions[dir].dir[k] * (I * dir_grid->directions[dir].area);
        }
    }
  }

  for (int h = 0; h < CELL_SIZE; h++)
    Impuls[h] /= dir_grid->full_area;

  return;
}

#endif // USE_CUDA