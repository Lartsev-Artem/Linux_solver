
#include <hip/hip_runtime.h>
#ifdef USE_CUDA
#include "cuda_illum_param.h"
#include "cuda_integrator.h"

#include "global_def.h"

#ifndef TRANSFER_CELL_TO_FACE

#ifdef ON_FULL_ILLUM_ARRAYS
#define CUDA_CONVERT_FACE_TO_CELL(val, size, src) \
  for (int k = 0; k < size; k++) {                \
    val(k) = 0;                                   \
    for (int f = 0; f < CELL_SIZE; f++)           \
      val(k) += src[f][k];                        \
    val(k) /= CELL_SIZE;                          \
  }

__device__ void cuda::device::MakeEnergy(const geo::grid_directions_device_t *dir, geo::grid_device_t *grid) {
  const IdType N = grid->loc_size;
  const IdType shift = grid->shift;
  const IdType i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= N)
    return;

  grid->energy[i] = direction_integrator::IntegrateByCell(shift + i, dir, grid);
}
#endif // ON_FULL_ILLUM_ARRAYS

__device__ void cuda::device::MakeDivStream(const geo::grid_directions_device_t *dir, geo::grid_device_t *grid) {
  //  const int M = dir->size;
  const IdType N = grid->loc_size;
  const IdType shift = grid->shift;

  IdType i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= N)
    return;

  Vector3 Stream[CELL_SIZE];
  direction_integrator::IntegrateByFaces3(i + shift, dir, grid, Stream);

#ifdef ON_FULL_ILLUM_ARRAYS
  CUDA_CONVERT_FACE_TO_CELL(grid->stream[i], 3, Stream);
#endif

  grid->divstream[i] = 0;
  IdType pos = (i + shift) * CELL_SIZE;
  Type div = 0;
  for (int f = 0; f < CELL_SIZE; f++) {
    Type sum = 0;
    for (int k = 0; k < 3; k++) {
      sum += Stream[f][k] * grid->normals[pos + f][k];
    }
    div += sum * grid->areas[pos + f];
  }

  grid->divstream[i] = div / grid->volume[i + shift];
  return;
}

__device__ void cuda::device::MakeDivImpuls(const geo::grid_directions_device_t *dir, geo::grid_device_t *grid) {
  // const int M = dir->size;
  const IdType N = grid->loc_size;
  const IdType shift = grid->shift;

  const IdType i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= N)
    return;

  Matrix3 impuls[CELL_SIZE];
  direction_integrator::IntegrateByFaces9(i + shift, dir, grid, impuls);

#ifdef ON_FULL_ILLUM_ARRAYS
  // CUDA_CONVERT_FACE_TO_CELL(grid->impuls[i], 9, impuls);
  grid->impuls[i] = Matrix3::Zero();
  for (int f = 0; f < CELL_SIZE; f++)
    grid->impuls[i] += impuls[f];
  grid->impuls[i] /= CELL_SIZE;
#endif

  Vector3 div = Vector3::Zero();

  for (IdType j = 0; j < CELL_SIZE; j++) {
    IdType pos = (i + shift) * CELL_SIZE + j;
    for (int h = 0; h < 3; h++) {
      Type sum = 0;
      for (int k = 0; k < 3; k++) {
        sum += impuls[j](h, k) * grid->normals[pos][k];
      }

      div[h] += sum * grid->areas[pos];
    }
  }

  grid->divimpuls[i] = div / grid->volume[i + shift];
  return;
}

#undef CUDA_CONVERT_FACE_TO_CELL

__global__ void cuda::kernel::MakeIllumParam(const cuda::geo::grid_directions_device_t *dir, cuda::geo::grid_device_t *grid) {
  // эти функции можно объденить в одну. Тогда будет одно общее обращение в память к illum
  device::MakeEnergy(dir, grid);
  device::MakeDivStream(dir, grid);
  device::MakeDivImpuls(dir, grid);
}

#else
__device__ void cuda::device::MakeEnergy(const geo::grid_directions_device_t *dir, geo::grid_device_t *grid) {

  const IdType i = blockIdx.x * blockDim.x + threadIdx.x;
  const IdType N = grid->loc_size;
  const IdType M = dir->size;

  if (i >= N)
    return;

  Type sum = 0;
  for (IdType k = 0; k < M; k++) {
    sum += grid->illum[M * i + k] * dir->directions[k].area;
  }

  grid->energy[i] = sum / dir->full_area; // direction_integrator::IntegrateByCell(shift + i, dir, grid);
}

///\todo наполнение функций
__device__ void cuda::device::MakeStream(const geo::grid_directions_device_t *dir, geo::grid_device_t *grid) {
}
__device__ void cuda::device::MakeImpuls(const geo::grid_directions_device_t *dir, geo::grid_device_t *grid) {
}

/// \note если данные на ячейках, то дивергенции на прямую не посчитать (для rad_rhd они не нужны)
__global__ void cuda::kernel::MakeIllumParam(const cuda::geo::grid_directions_device_t *dir, cuda::geo::grid_device_t *grid) {
  // эти функции можно объденить в одну. Тогда будет одно общее обращение в память к illum
  device::MakeEnergy(dir, grid);
  device::MakeStream(dir, grid);
  device::MakeImpuls(dir, grid);
}
#endif //! TRANSFER_CELL_TO_FACE

#endif //! USE_CUDA