#include "hip/hip_runtime.h"
#include "cuda_init_mem.h"
#include "cuda_memory.h"
#include "cuda_multi_init.h"
#include "cuda_multi_interface.h"

#include "mpi_shifts.h"

#ifdef SEPARATE_GPU
cuda::multi_gpu_config_t gpu_config;

std::vector<cuda::geo::grid_directions_device_t *> grid_dir_deviceN;
std::vector<cuda::geo::grid_device_t *> grid_deviceN;
std::vector<cuda::geo::device_host_ptr_t> device_host_ptrN;

static void init_params_config(cuda::multi_gpu_config_t &gpu_conf, const grid_t &grid_host) {

  gpu_conf.size_params.resize(gpu_conf.GPU_N, 0);
  gpu_conf.disp_params.resize(gpu_conf.GPU_N, 0);

  const IdType loc_size_grid = grid_host.loc_size;
  const IdType loc_shift_grid = grid_host.loc_shift;

  for (size_t id_dev = 0; id_dev < gpu_conf.GPU_N; id_dev++) {

    const IdType cells_loc = gpu_conf.size[id_dev];
    const IdType cells_shift = gpu_conf.disp[id_dev];

    //левая граница на узле больше правой на карте
    if (loc_shift_grid > cells_shift + cells_loc) {
      continue;
    }

    //правая граница на узле меньше левой на карте
    if (loc_shift_grid + loc_size_grid < cells_shift) {
      continue;
    }

    //здесь уже точно есть пересечение
    int left = std::max(loc_shift_grid, cells_shift);
    int right = std::min(loc_shift_grid + loc_size_grid, cells_shift + cells_loc);

    gpu_conf.disp_params[id_dev] = left;
    gpu_conf.size_params[id_dev] = right - left;
  }
}

int cuda::interface::separate_device::InitDevice(const grid_directions_t &grid_dir_host, grid_t &grid_host) {

  CUDA_CALL_FUNC(hipGetDeviceCount, &gpu_config.GPU_N);
#ifdef SINGLE_GPU
  gpu_config.GPU_N = GPU_DIV_PARAM;
#endif
  GetSend(gpu_config.GPU_N, grid_host.size, gpu_config.size);
  GetDisp(gpu_config.GPU_N, grid_host.size, gpu_config.disp);
  init_params_config(gpu_config, grid_host);

  int _dev_n = gpu_config.GPU_N; //реальное число карт
#ifdef SINGLE_GPU
  _dev_n = 1;
#endif

  device_host_ptrN.resize(_dev_n);
  grid_dir_deviceN.resize(_dev_n);
  grid_deviceN.resize(_dev_n);

  for (int dev_id = 0; dev_id < _dev_n; dev_id++) {
    CUDA_CALL_FUNC(hipSetDevice, dev_id);
    cuda::separate_device::InitDirectionsOnMultiDevice(grid_dir_host, device_host_ptrN[dev_id], grid_dir_deviceN[dev_id]);
    cuda::separate_device::InitMultiDeviceGrid(dev_id, gpu_config, grid_host, grid_dir_host, device_host_ptrN[dev_id], grid_deviceN[dev_id]);
  }

  /// \todo это отдельно, т.к. относится к инициализации хоста
  mem_protected::MallocHost((grid_dir_host.size * grid_host.size * sizeof(Type)), &grid_host.Illum);
  mem_protected::MallocHost(((grid_dir_host.loc_size) * grid_host.size * sizeof(Type)), &grid_host.scattering);

  WRITE_LOG("grid_host.loc_size =%lu %lu %lu %lu\n", grid_host.loc_size, grid_host.loc_shift, grid_host.size, grid_host.size_face);

  if (grid_dir_host.loc_shift == 0 || (grid_host.size != grid_host.loc_size)) // или нулевой узел, или данные разделены
  {
    mem_protected::MallocHost((grid_host.loc_size * sizeof(Type)), &grid_host.energy);
    mem_protected::MallocHost((grid_host.loc_size * sizeof(Vector3)), &grid_host.stream);
    mem_protected::MallocHost((grid_host.loc_size * sizeof(Matrix3)), &grid_host.impuls);

    // //сопоставимо с хранением energy и stream и impuls
    // const Type abs_op = grid.cells[cell].illum_val.absorp_coef;
    // const Type scat_op = grid.cells[cell].illum_val.scat_coef;
    // const Type rho = grid.cells[cell].phys_val.d;
    // const Type prs = grid.cells[cell].phys_val.p;
    // const Vector3 &v = grid.cells[cell].phys_val.v;
  }

  return e_completion_success;
}

void cuda::interface::separate_device::ClearDevice() {

  cuda::separate_device::ClearDirectionsOnMultiDevice(gpu_config, device_host_ptrN, grid_dir_deviceN);
  cuda::separate_device::ClearGridOnMultiDevice(gpu_config, device_host_ptrN, grid_deviceN);

  // CUDA_CALL_FUNC(hipDeviceReset);

  WRITE_LOG("Free device arrays\n");
}
void cuda::interface::separate_device::ClearHost(grid_t &grid_host) {

  mem_protected::FreeMemHost(grid_host.Illum);
  mem_protected::FreeMemHost(grid_host.scattering);

#ifdef ON_FULL_ILLUM_ARRAYS
  mem_protected::FreeMemHost(grid_host.energy);
  mem_protected::FreeMemHost(grid_host.stream);
  mem_protected::FreeMemHost(grid_host.impuls);
#endif

  WRITE_LOG("Free host arrays\n");
}

#endif //! SEPARATE_GPU
