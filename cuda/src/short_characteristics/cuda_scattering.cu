
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "cuda_integrator.h"
#include "cuda_scattering.h"
#include "global_def.h"

__global__ void cuda::kernel::GetS(const geo::grid_directions_device_t *dir, geo::grid_device_t *grid) {
  const int M = dir->size;
  const int N = grid->size;

  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int k = blockIdx.y * blockDim.y + threadIdx.y;

  if (i >= N || k >= M)
    return;

  const Vector3 &cur_dir = dir->directions[k].dir;
  const Type *Illum = grid->illum;
  const geo::direction_device_t *all_dir = dir->directions;

  Type scatter = 0;
  for (int num_direction = 0; num_direction < M; num_direction++) {
    int pos = CELL_SIZE * (num_direction * N + i);
    Type I = (Illum[pos] + Illum[pos + 1] + Illum[pos + 2] + Illum[pos + 3]) / 4.;
    scatter += direction_integrator::Gamma(all_dir[num_direction].dir, cur_dir) * I * all_dir[num_direction].area;
  }

  grid->int_scattering[k * N + i] = scatter / dir->full_area;
}

__global__ void cuda::kernel::GetS_MPI(const geo::grid_directions_device_t *dir, geo::grid_device_t *grid) {
  const int N = grid->size;

  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int k = blockIdx.y * blockDim.y + threadIdx.y;

  if (i >= N || k >= grid->local_scattering_size)
    return;

  const int M = dir->size;

  const Vector3 &cur_dir = dir->directions[grid->local_scattering_disp + k].dir;
  const Type *Illum = grid->illum;
  const geo::direction_device_t *all_dir = dir->directions;

  Type scatter = 0;
  for (int num_direction = 0; num_direction < M; num_direction++) {
    int pos = CELL_SIZE * (num_direction * N + i);
    Type I = (Illum[pos] + Illum[pos + 1] + Illum[pos + 2] + Illum[pos + 3]) / 4.;
    scatter += direction_integrator::Gamma(all_dir[num_direction].dir, cur_dir) * I * all_dir[num_direction].area;
  }

  grid->int_scattering[k * N + i] = scatter / dir->full_area;
}

__global__ void cuda::kernel::GetS_MPI_Stream(const geo::grid_directions_device_t *dir, geo::grid_device_t *grid, const int start, const int end) {
  const int N = grid->size;

  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int k = blockIdx.y * blockDim.y + threadIdx.y;

  if (i >= N || k >= end || k < start)
    return;

  const int M = dir->size;

  const Vector3 &cur_dir = dir->directions[grid->local_scattering_disp + k].dir;
  const Type *Illum = grid->illum;
  const geo::direction_device_t *all_dir = dir->directions;

  Type scatter = 0;
  for (int num_direction = 0; num_direction < M; num_direction++) {
    int pos = CELL_SIZE * (num_direction * N + i);
    Type I = (Illum[pos] + Illum[pos + 1] + Illum[pos + 2] + Illum[pos + 3]) / 4.;
    scatter += direction_integrator::Gamma(all_dir[num_direction].dir, cur_dir) * I * all_dir[num_direction].area;
  }

  grid->int_scattering[k * N + i] = scatter / dir->full_area;
}

#endif //! USE_CUDA